#include "hip/hip_runtime.h"
#include <cassert>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>

#include "lbp.cuh"

__global__ void lbp_value_kernel(const unsigned char* image,
                                 unsigned char* lbp_values, const int width,
                                 const int height, const size_t pitch)
{
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;

    if (x >= width || y >= height)
        return;

    auto pixel_value = image[y * width + x];

    unsigned char lbp_value = 0;

    lbp_value |= (threadIdx.y > 0
                    && image[(y - 1) * width + x] >= pixel_value) << 7;
    lbp_value |= (threadIdx.y > 0 && threadIdx.x > 0
                    && image[(y - 1) * width + x - 1] >= pixel_value) << 6;
    lbp_value |= (threadIdx.x > 0
                    && image[y * width + x - 1] >= pixel_value) << 5;
    lbp_value |= (threadIdx.y < 15 && threadIdx.x > 0
                    && image[(y + 1) * width + x - 1] >= pixel_value) << 4;
    lbp_value |= (threadIdx.y < 15
                    && image[(y + 1) * width + x] >= pixel_value) << 3;
    lbp_value |= (threadIdx.y < 15 && threadIdx.x < 15
                    && image[(y + 1) * width + x + 1] >= pixel_value) << 2;
    lbp_value |= (threadIdx.x < 15
                    && image[y * width + x + 1] >= pixel_value) << 1;
    lbp_value |= threadIdx.y > 0 && threadIdx.x < 15
                    && image[(y - 1) * width + x + 1] >= pixel_value;

    auto tile_index = blockIdx.y * width / 16 + blockIdx.x;
    auto pixel_index_in_tile = blockDim.x * threadIdx.y + threadIdx.x;

    auto lbp_index = lbp_values + tile_index * pitch
        + pixel_index_in_tile * sizeof(unsigned char);
    *lbp_index = lbp_value;

    // printf("(%d, %d) = %u\n", x, y, lbp_value);
}

__global__ void compute_histo_kernel(unsigned char *histo_tab, const unsigned char* lbp_values)
{
    
}

void compute_lbp_values(const unsigned char* image, const size_t width,
                        const size_t height)
{
    hipError_t rc = hipSuccess;

    unsigned char* cuda_image;
    auto pixels_number = width * height;

    rc = hipMalloc(&cuda_image, pixels_number * sizeof(unsigned char));
    if (rc)
    {
        std::cout << "Could not allocate memory for the image on the device\n";
        exit(EXIT_FAILURE);
    }

    rc = hipMemcpy(cuda_image, image, pixels_number * sizeof(unsigned char),
                    hipMemcpyHostToDevice);
    if (rc)
    {
        std::cout << "Could not copy image data from host to device\n";
        exit(EXIT_FAILURE);
    }

    unsigned char* lbp_values;
    size_t lbp_pitch;
    auto tiles_number = width * height / 256;

    rc = hipMallocPitch(&lbp_values, &lbp_pitch, 256 * sizeof(unsigned char), tiles_number);
    if (rc)
    {
        std::cout << "Could not allocate memory for lbp values buffer\n";
        exit(EXIT_FAILURE);
    }

    int bsize = 16;
    int w = std::ceil((float)width / bsize);
    int h = std::ceil((float)height / bsize);

    /*std::cout << "Running kernel of size ("
              << w << ", " << h << ")\n";*/

    dim3 lbp_dim_block(bsize, bsize);
    dim3 lbp_dim_grid(w, h);

    lbp_value_kernel<<<lbp_dim_grid, lbp_dim_block>>>(cuda_image, lbp_values, width, height, lbp_pitch);

    hipDeviceSynchronize();

    unsigned char* histo_tab;
    size_t histo_pitch;
    rc = hipMallocPitch(&histo_tab, &histo_pitch, 256 * sizeof(unsigned char), tiles_number);
    if (rc)
    {
        std::cout << "Could not allocate memory for lbp values buffer\n";
        exit(EXIT_FAILURE);
    }

    hipMemset2D(histo_tab, histo_pitch, 0, 256, tiles_number);

    /*std::cout << (int) image[0] << " " << (int) image[1] << " " << (int)
    image[2] << "\n"
                << (int) image[width] << " " << (int) image[width + 1] << "
    " << (int) image[width + 2] << "\n"
                << (int) image[2 * width] << " " << (int) image[2 * width +
    1]
    << " " << (int) image[2 * width + 2] << "\n";
    */


    unsigned char* output = (unsigned char*) malloc(256 * sizeof(unsigned char));
    if (output == NULL)
    {
        std::cout << "CRINGE\n";
        exit(EXIT_FAILURE);
    }

    hipMemcpy2D(output, 0, histo_tab, histo_pitch, 256 * sizeof(unsigned char), 1, hipMemcpyDeviceToHost);

    for (auto i = 0; i < 256; i++)
        std::cout << i << ": " << (int) *(output + i) << "\n";

    hipFree(cuda_image);
    hipFree(lbp_values);
    hipFree(histo_tab);
}
